
#include <hip/hip_runtime.h>
#include <array>
#include <chrono>
#include <cstdint>
#include <iostream>

constexpr auto per_thread = 2 * 32768ULL;
constexpr auto per_iter = per_thread * 32768ULL;

__global__ void run(uint64_t base, unsigned long long* result)
{
    constexpr std::array<uint64_t, 16> prog = {2, 4, 1, 1, 7, 5, 0, 3, 4, 3, 1, 6, 5, 5, 3, 0};

    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (auto offset = 0ULL; offset < per_thread; ++offset) {
        uint64_t const a_init = base + idx * per_thread + offset;
        uint64_t a = base + idx * per_thread + offset;

        for (auto i = 0; a > 0; ++i, a >>= 3) {
            uint64_t b = b = (a & 7) ^ 1;
            b = (b ^ (a >> b)) ^ 6;

            if ((b & 7) != prog[i]) break;

            if (i + 1 == prog.size()) {
                atomicMin(result, a_init);
                return;
            }
        }
    }
}

#define CHECK(code)                                                                      \
    if ((code) != hipSuccess) {                                                         \
        std::cerr << "CUDA call failed at " << __FILE__ << ":" << __LINE__ << std::endl; \
        std::terminate();                                                                \
    }

int main()
{
    constexpr auto num_threads = 256;
    constexpr auto num_blocks = per_iter / per_thread / num_threads;

    unsigned long long* result_gpu = nullptr;
    CHECK(hipMalloc(&result_gpu, sizeof(uint64_t)));

    uint64_t result = ~0;
    CHECK(hipMemcpy(result_gpu, &result, sizeof(result), hipMemcpyHostToDevice));

    auto begin = std::chrono::high_resolution_clock::now();

    constexpr auto iters_between_status = 1000ULL;

    for (uint64_t base = 35184372088832, i = 0;; base += per_iter, ++i) {
        run<<<num_blocks, num_threads>>>(base, result_gpu);
        uint64_t result;
        CHECK(hipMemcpy(&result, result_gpu, sizeof(result), hipMemcpyDeviceToHost));
        if (result != ~0) {
            std::cout << "Part 2: " << result << std::endl;
            break;
        }
        if ((i + 1) % iters_between_status == 0) {
            auto const done = (i + 1) * per_iter;
            auto const now = std::chrono::high_resolution_clock::now();
            std::cout << "base " << base << std::endl;
            std::cout << done << " values tried ("
                      << static_cast<double>(per_iter * iters_between_status) /
                             std::chrono::duration_cast<std::chrono::microseconds>(now - begin).count() / 1e3
                      << " * 10^9 it/s)" << std::endl;
            begin = now;
        }
    }

    return 0;
}
